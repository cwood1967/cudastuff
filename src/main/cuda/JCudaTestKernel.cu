extern "C"

#include <hip/hip_runtime.h>
#include <math.h>

__global__ void test(int n, float *a, float*b, float *res)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n)
    {
        res[i] = exp(-a[i]*b[i]);
    }
}        